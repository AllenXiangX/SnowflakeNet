#include "hip/hip_runtime.h"
// EMD approximation module (based on auction algorithm)
// author: Minghua Liu
#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

__device__ __forceinline__ float atomicMax(float *address, float val)
{
    int ret = __float_as_int(*address);
    while(val > __int_as_float(ret))
    {
        int old = ret;
        if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
            break;
    }
    return __int_as_float(ret);
}


__global__ void clear(int b, int * cnt_tmp, int * unass_cnt) {
	for (int i = threadIdx.x; i < b; i += blockDim.x) {
		cnt_tmp[i] = 0;
		unass_cnt[i] = 0;
	}
}

__global__ void calc_unass_cnt(int b, int n, int * assignment, int * unass_cnt) { 
	// count the number of unassigned points in each batch
	const int BLOCK_SIZE = 256; 
	__shared__ int scan_array[BLOCK_SIZE];
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		scan_array[threadIdx.x] = assignment[i * n + blockIdx.y * BLOCK_SIZE + threadIdx.x] == -1 ? 1 : 0;
		__syncthreads();
		
		int stride = 1;
		while(stride <= BLOCK_SIZE / 2) {
			int index = (threadIdx.x + 1) * stride * 2 - 1; 
			if(index < BLOCK_SIZE)
				scan_array[index] += scan_array[index - stride]; 
			stride = stride * 2;
			__syncthreads(); 
		}
		__syncthreads();
		
		if (threadIdx.x == BLOCK_SIZE - 1) {
			atomicAdd(&unass_cnt[i], scan_array[threadIdx.x]);
		}
		__syncthreads();
	}
}

__global__ void calc_unass_cnt_sum(int b, int * unass_cnt, int * unass_cnt_sum) {
	// count the cumulative sum over over unass_cnt
	const int BLOCK_SIZE = 512; // batch_size <= 512
	__shared__ int scan_array[BLOCK_SIZE];
	scan_array[threadIdx.x] = unass_cnt[threadIdx.x];
	__syncthreads();
	
	int stride = 1;
	while(stride <= BLOCK_SIZE / 2) {
		int index = (threadIdx.x + 1) * stride * 2 - 1; 
		if(index < BLOCK_SIZE)
			scan_array[index] += scan_array[index - stride]; 
		stride = stride * 2;
		__syncthreads(); 
	}
	__syncthreads();
	stride = BLOCK_SIZE / 4; 
	while(stride > 0) {
		int index = (threadIdx.x + 1) * stride * 2 - 1; 
		if((index + stride) < BLOCK_SIZE)
			scan_array[index + stride] += scan_array[index];
		stride = stride / 2;
		__syncthreads(); 
	}
	__syncthreads(); 
	
	//printf("%d\n", unass_cnt_sum[b - 1]);
	unass_cnt_sum[threadIdx.x] = scan_array[threadIdx.x];
}

__global__ void calc_unass_idx(int b, int n, int * assignment, int * unass_idx, int * unass_cnt, int * unass_cnt_sum, int * cnt_tmp) {
	// list all the unassigned points
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		if (assignment[i * n + blockIdx.y * 256 + threadIdx.x] == -1) {
			int idx = atomicAdd(&cnt_tmp[i], 1);
			unass_idx[unass_cnt_sum[i] - unass_cnt[i] + idx] = blockIdx.y * 256 + threadIdx.x;
		} 
	}
}

__global__ void Bid(int b, int n, const float * xyz1, const float * xyz2, float eps, int * assignment, int * assignment_inv, float * price, 
					int * bid, float * bid_increments, float * max_increments, int * unass_cnt, int * unass_cnt_sum, int * unass_idx) {
	const int batch = 2048, block_size = 256, block_cnt = n / 256;
	__shared__ float xyz2_buf[batch * 3];
	__shared__ float price_buf[batch];
	__shared__ float best_buf[block_size];
	__shared__ float better_buf[block_size];
	__shared__ int best_i_buf[block_size];
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		int _unass_cnt = unass_cnt[i];
		if (_unass_cnt == 0)
			continue;
		int _unass_cnt_sum = unass_cnt_sum[i];
		int unass_per_block = (_unass_cnt + block_cnt - 1) / block_cnt;
		int thread_per_unass = block_size / unass_per_block;
		int unass_this_block = max(min(_unass_cnt - (int) blockIdx.y * unass_per_block, unass_per_block), 0);
			
		float x1, y1, z1, best = -1e9, better = -1e9;
		int best_i = -1, _unass_id = -1, thread_in_unass;

		if (threadIdx.x < thread_per_unass * unass_this_block) {
			_unass_id = unass_per_block * blockIdx.y + threadIdx.x / thread_per_unass + _unass_cnt_sum - _unass_cnt;
			_unass_id = unass_idx[_unass_id];
			thread_in_unass = threadIdx.x % thread_per_unass;

			x1 = xyz1[(i * n + _unass_id) * 3 + 0];
			y1 = xyz1[(i * n + _unass_id) * 3 + 1];
			z1 = xyz1[(i * n + _unass_id) * 3 + 2];
		}

		for (int k2 = 0; k2 < n; k2 += batch) {
			int end_k = min(n, k2 + batch) - k2;
			for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
				xyz2_buf[j] = xyz2[(i * n + k2) * 3 + j];
			}
			for (int j = threadIdx.x; j < end_k; j += blockDim.x) {
				price_buf[j] = price[i * n + k2 + j];
			}
			__syncthreads();

			if (_unass_id != -1) {
				int delta = (end_k + thread_per_unass - 1) / thread_per_unass;
				int l = thread_in_unass * delta;
				int r = min((thread_in_unass + 1) * delta, end_k);
				for (int k = l; k < r; k++) 
				//if (!last || assignment_inv[i * n + k + k2] == -1)
				{
					float x2 = xyz2_buf[k * 3 + 0] - x1;
					float y2 = xyz2_buf[k * 3 + 1] - y1;
					float z2 = xyz2_buf[k * 3 + 2] - z1;
					// the coordinates of points should be normalized to [0, 1]
					float d = 3.0 - sqrtf(x2 * x2 + y2 * y2 + z2 * z2) - price_buf[k];
					if (d > best) {
						better = best;
						best = d;
						best_i = k + k2;
					}
					else if (d > better) {
						better = d;
					}
				}
			}
			__syncthreads();
		}

		best_buf[threadIdx.x] = best;
		better_buf[threadIdx.x] = better;
		best_i_buf[threadIdx.x] = best_i;
		__syncthreads();
		
		if (_unass_id != -1 && thread_in_unass == 0) {
			for (int j = threadIdx.x + 1; j < threadIdx.x + thread_per_unass; j++) {
				if (best_buf[j] > best) {
					better = max(best, better_buf[j]);
					best = best_buf[j];
					best_i = best_i_buf[j];
				}
				else better = max(better, best_buf[j]);
			}
			bid[i * n + _unass_id] = best_i;
			bid_increments[i * n + _unass_id] = best - better + eps; 
			atomicMax(&max_increments[i * n + best_i], best - better + eps);
		}
	}
}

__global__ void GetMax(int b, int n, int * assignment, int * bid, float * bid_increments, float * max_increments, int * max_idx) {
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		int j = threadIdx.x + blockIdx.y * blockDim.x;
		if (assignment[i * n + j] == -1) {
			int bid_id = bid[i * n + j];
			float bid_inc = bid_increments[i * n + j];
			float max_inc = max_increments[i * n + bid_id];
			if (bid_inc - 1e-6 <= max_inc && max_inc <= bid_inc + 1e-6) 
			{
				max_idx[i * n + bid_id] = j;
			}
		}
	}
}

__global__ void Assign(int b, int n, int * assignment, int * assignment_inv, float * price, int * bid, float * bid_increments, float * max_increments, int * max_idx, bool last) {
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		int j = threadIdx.x + blockIdx.y * blockDim.x;
		if (assignment[i * n + j] == -1) {
			int bid_id = bid[i * n + j];
			if (last || max_idx[i * n + bid_id] == j) 
			{
				float bid_inc = bid_increments[i * n + j];
				int ass_inv = assignment_inv[i * n + bid_id];
				if (!last && ass_inv != -1) {
					assignment[i * n + ass_inv] = -1;
				}
				assignment_inv[i * n + bid_id] = j;
				assignment[i * n + j] = bid_id;
				price[i * n + bid_id] += bid_inc;
				max_increments[i * n + bid_id] = -1e9;
			}
		}
	}
}

__global__ void CalcDist(int b, int n, float * xyz1, float * xyz2, float * dist, int * assignment) {
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		int j = threadIdx.x + blockIdx.y * blockDim.x;
		int k = assignment[i * n + j];
		float deltax = xyz1[(i * n + j) * 3 + 0] - xyz2[(i * n + k) * 3 + 0];
		float deltay = xyz1[(i * n + j) * 3 + 1] - xyz2[(i * n + k) * 3 + 1];
		float deltaz = xyz1[(i * n + j) * 3 + 2] - xyz2[(i * n + k) * 3 + 2];
		dist[i * n + j] = deltax * deltax + deltay * deltay + deltaz * deltaz;
	}
}

int emd_cuda_forward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor dist, at::Tensor assignment, at::Tensor price, 
	                 at::Tensor assignment_inv, at::Tensor bid, at::Tensor bid_increments, at::Tensor max_increments,
	                 at::Tensor unass_idx, at::Tensor unass_cnt, at::Tensor unass_cnt_sum, at::Tensor cnt_tmp, at::Tensor max_idx, float eps, int iters) {

	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B
	
	if (n != m) {
		printf("Input Error! The two point clouds should have the same size.\n");
		return -1;
	}

	if (batch_size > 512) {
		printf("Input Error! The batch size should be less than 512.\n");
		return -1;
	}

	if (n % 256 != 0) {
		printf("Input Error! The size of the point clouds should be a multiple of 256.\n");
		return -1;
	}

	//hipEvent_t start,stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//hipEventRecord(start);
	//int iters = 50;
	for (int i = 0; i < iters; i++) {
		clear<<<1, batch_size>>>(batch_size, cnt_tmp.data<int>(), unass_cnt.data<int>());
		calc_unass_cnt<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, assignment.data<int>(), unass_cnt.data<int>());
		calc_unass_cnt_sum<<<1, batch_size>>>(batch_size, unass_cnt.data<int>(), unass_cnt_sum.data<int>());
		calc_unass_idx<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, assignment.data<int>(), unass_idx.data<int>(), unass_cnt.data<int>(), 
											 unass_cnt_sum.data<int>(), cnt_tmp.data<int>());
		Bid<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, xyz1.data<float>(), xyz2.data<float>(), eps, assignment.data<int>(), assignment_inv.data<int>(), 
			                          price.data<float>(), bid.data<int>(), bid_increments.data<float>(), max_increments.data<float>(),
			                          unass_cnt.data<int>(), unass_cnt_sum.data<int>(), unass_idx.data<int>());
		GetMax<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, assignment.data<int>(), bid.data<int>(), bid_increments.data<float>(), max_increments.data<float>(), max_idx.data<int>());
		Assign<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, assignment.data<int>(), assignment_inv.data<int>(), price.data<float>(), bid.data<int>(),
									  bid_increments.data<float>(), max_increments.data<float>(), max_idx.data<int>(), i == iters - 1);
	}
	CalcDist<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, xyz1.data<float>(), xyz2.data<float>(), dist.data<float>(), assignment.data<int>());
	//hipEventRecord(stop);
	//hipEventSynchronize(stop);
	//float elapsedTime;
	//hipEventElapsedTime(&elapsedTime,start,stop);
	//printf("%lf\n", elapsedTime);

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd Output: %s\n", hipGetErrorString(err));
	    return 0;
	  }
	  return 1;
}

__global__ void NmDistanceGradKernel(int b, int n, const float * xyz1, const float * xyz2, const float * grad_dist, const int * idx, float * grad_xyz){
	for (int i = blockIdx.x; i < b; i += gridDim.x) {
		for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
			float x1 = xyz1[(i * n + j) * 3 + 0];
			float y1 = xyz1[(i * n + j) * 3 + 1];
			float z1 = xyz1[(i * n + j) * 3 + 2];
			int j2 = idx[i * n + j];
			float x2 = xyz2[(i * n + j2) * 3 + 0];
			float y2 = xyz2[(i * n + j2) * 3 + 1];
			float z2 = xyz2[(i * n + j2) * 3 + 2];
			float g = grad_dist[i * n + j] * 2;
			atomicAdd(&(grad_xyz[(i * n + j) * 3 + 0]), g * (x1 - x2));
			atomicAdd(&(grad_xyz[(i * n + j) * 3 + 1]), g * (y1 - y2));
			atomicAdd(&(grad_xyz[(i * n + j) * 3 + 2]), g * (z1 - z2));
		}
	}
}

int emd_cuda_backward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor gradxyz, at::Tensor graddist, at::Tensor idx){
	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); 
	const auto m = xyz2.size(1); 

	NmDistanceGradKernel<<<dim3(batch_size, n / 256, 1), 256>>>(batch_size, n, xyz1.data<float>(), xyz2.data<float>(), graddist.data<float>(), idx.data<int>(), gradxyz.data<float>());
	
	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd get grad: %s\n", hipGetErrorString(err));
	    return 0;
	  }
	  return 1;
	
}
